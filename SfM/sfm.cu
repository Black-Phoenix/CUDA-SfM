#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "kernels.h"
#include "sfm.h"
#include <assert.h>
#include <hipblas.h>
#include <hiprand.h>
#include <memory>
#include <string>
#include <iostream>
#include <fstream>
#include <hipblas.h>
#include <CudaSift/cudaSift.h>
#include <hiprand.h>
#include <iomanip>      
#include <random>
#include <algorithm>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <hipsolver.h>
#include "svd.h"


namespace SfM {

	Image_pair::Image_pair(float k[9], float k_inv[9], int image_count, int num_points) :image_count(image_count), num_points(num_points) { // num_points should be a array if we want to deal with more than 2 images
		hipMalloc((void**)&d_K, 3 * 3 * sizeof(float));
		checkCUDAErrorWithLine("Malloc failed!");
		hipMalloc((void**)&d_K_inv, 3 * 3 * sizeof(float));
		checkCUDAErrorWithLine("Malloc failed!");
		hipMemcpy(d_K, k, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_K_inv, k_inv, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
		// Allocate Point space
		float *d_u, *d_x;
		for (int i = 0; i < image_count; i++) {
			hipMalloc((void**)&d_u, 3 * num_points * sizeof(float));
			U.push_back(d_u);
			hipMalloc((void**)&d_x, 3 * num_points * sizeof(float));
			X.push_back(d_x);
		}
		// E
		hipMalloc((void **)&d_E, 3 * 3 * sizeof(float));
		// Canidate R, T
		hipMalloc((void **)&d_P, 4 * 4 * 4 * sizeof(float));
		// uniform svd handles
		float residual = 0;
		int executed_sweeps = 0;
		const float tol = 1.e-7;
		const int max_sweeps = 15;
		const int sort_svd = 1;
		using namespace std;
		hipsolverDnCreate(&cusolverH);
		hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
		checkCUDAError("Could not create flags");
		hipsolverSetStream(cusolverH, stream);
		checkCUDAError("Could not Set strea,");
		hipsolverDnCreateGesvdjInfo(&gesvdj_params);
		checkCUDAError("Could not create GesvdjInfo");
		hipsolverDnXgesvdjSetTolerance(
			gesvdj_params,
			tol);
		checkCUDAError("Could not SetTolerance");
		hipsolverDnXgesvdjSetMaxSweeps(
			gesvdj_params,
			max_sweeps);
		checkCUDAError("Could not SetMaxSweeps");
		hipsolverDnXgesvdjSetSortEig(
			gesvdj_params,
			sort_svd);
		checkCUDAError("Could not SetSortEigs");

		// cublas handle
		hipblasCreate(&handle);
		// Space for final points
		hipMalloc((void **)&d_final_points, 4 * num_points * sizeof(float));
	}

	void Image_pair::estimateE() {
		const int ransac_count = floor(num_points / 8);
		// Create random order of points (on cpu using std::shuffle)
		int *indices = new int[num_points];
		int *d_indices;
		hipMalloc((void **)&d_indices, num_points * sizeof(int));
		for (int i = 0; i < num_points; indices[i] = i, i++);
		// Shufle data
		std::random_device rd;
		std::mt19937 g(rd());
		//shuffle(indices, indices + num_points, g); todo enable this 
		// Copy data to gpu
		hipMemcpy(d_indices, indices, num_points * sizeof(int), hipMemcpyHostToDevice);
		// Calculate all kron products correctly
		float *d_A;
		hipMalloc((void **)&d_A, 8 * 9 * ransac_count * sizeof(float));
		checkCUDAErrorWithLine("A malloc failed!");
		int grids = ceil((ransac_count + cuda_block_size - 1) / cuda_block_size);
		kernels::kron_kernal << <grids, cuda_block_size >> > (X[0], X[1], d_A, d_indices, ransac_count, num_points);
		checkCUDAErrorWithLine("Kron failed!");
		
		float *d_E_canidate;
		hipMalloc((void **)&d_E_canidate, 3 * 3 * ransac_count * sizeof(float));
		// Calculate batch SVD of d_A
		float *d_ut, *d_vt, *d_s;
		hipMalloc((void **)&d_ut, 8 * 8 * ransac_count * sizeof(float));
		hipMalloc((void **)&d_vt, 9 * 9 * ransac_count * sizeof(float));
		hipMalloc((void **)&d_s, 8 * ransac_count * sizeof(float));
		int *d_info = NULL;
		hipMalloc((void**)&d_info, 4 * sizeof(int));
		kernels::regular_svd(d_A, d_ut, d_s, d_vt, 8, 9, ransac_count, d_info, cusolverH, gesvdj_params);
		// Last column of V becomes E (row of v' in our case)
		int blocks = ceil((ransac_count + cuda_block_size - 1) / cuda_block_size);
		kernels::row_extraction_kernel << <blocks, cuda_block_size >> > (d_vt, d_E_canidate, ransac_count);
		// Calculate target E's
		kernels::normalizeE << <grids, cuda_block_size >> > (d_E_canidate, ransac_count);
		
		// Calculate number of inliers for each E
		int *inliers = calculateInliers(d_E_canidate, ransac_count);
		kernels::printVector(inliers, ransac_count, "inliers");
		// Pick best E and allocate d_E and E using thrust
		thrust::device_ptr<int> dv_in(inliers);
		auto iter = thrust::max_element(dv_in, dv_in + ransac_count);
		unsigned int best_pos = (iter - dv_in) - 1;
		// Assigne d_E
		hipMemcpy(d_E, &(d_E_canidate[access3(0, 0, best_pos, 3, 3)]), 3 * 3 * sizeof(float), hipMemcpyDeviceToDevice);
		kernels::printMatrix(d_E, 3, 3, 3, "d_E!!!");
		// Free stuff
		hipFree(inliers);
		hipFree(d_A);
		// svd free
		hipFree(d_ut);
		hipFree(d_s);
		hipFree(d_vt);
		hipFree(d_info);
		hipFree(d_indices);
		free(indices);
		hipFree(d_E_canidate);
	}

	void Image_pair::fillXU(SiftPoint *data) {
		kernels::printMatrix(d_K_inv, 3, 3, 3, "K inv");
		// Fill U
		dim3 grids(ceil((num_points + cuda_block_size - 1) / cuda_block_size), 1);
		dim3 blocks(cuda_block_size, 3);
		kernels::copy_point << <grids, blocks >> > (data, num_points, U[0], U[1]);
		kernels::printMatrix(U[0], 3, num_points, 5, "U[0]");
		// Fill X using X = inv(K) * U
		kernels::gpu_blas_mmul(d_K_inv, U[0], X[0], 3, 3, num_points, handle);
		kernels::gpu_blas_mmul(d_K_inv, U[1], X[1], 3, 3, num_points, handle);
		kernels::printMatrix(X[0], 3, num_points, 2, "X[0]");
		kernels::printMatrix(X[1], 3, num_points, 2, "X[1]");
	}

	int * Image_pair::calculateInliers(float *d_E_canidate, int ransac_iter) {
		/// This function calculates n1, d1, n2, d2 and then finds the number of residuals per E canidate in X[0] and X[1]
		// Init E1
		float E1[9] = { 0, -1, 0, 1, 0, 0, 0, 0, 0 };
		float *d_E1;
		hipMalloc((void **)&d_E1, 9 * sizeof(float));
		hipMemcpy(d_E1, E1, 9 * sizeof(float), hipMemcpyHostToDevice);
		// Allocs
		float *x1_transformed, *x2_transformed;
		hipMalloc((void**)&x1_transformed, 3 * num_points * ransac_iter * sizeof(float));
		hipMalloc((void**)&x2_transformed, 3 * num_points * ransac_iter * sizeof(float));
		float *d1, *d2;
		hipMalloc((void**)&d1, 3 * num_points * ransac_iter * sizeof(float));
		hipMalloc((void**)&d2, 3 * num_points * ransac_iter * sizeof(float));
		float *n1, *n2;
		hipMalloc((void **)&n1, 3 * num_points * ransac_iter * sizeof(float));
		hipMalloc((void **)&n2, 3 * num_points * ransac_iter * sizeof(float));
		// Calculate x1 (from matlab code) {
		int m = 3, k = 3, n = num_points;
		kernels::gpu_blas_mmul_batched(d_E_canidate, X[0], x1_transformed, m, k, n, m * k, 0, m * n, ransac_iter, handle);

		//Compute n1 
		m = num_points, k = 3, n = 3; // these probably need to change because we need to transpose X[1]
		kernels::gpu_blas_mmul_transpose_batched(X[1], d_E_canidate, n1, m, k, n, 0, 3 * 3, m * n, ransac_iter, handle); // transpose X[1]
		int blocks = ceil((3 * num_points + cuda_block_size - 1) / cuda_block_size); // BUG!!! we need to make this batched
		kernels::element_wise_mult << <blocks, cuda_block_size >> > (n1, X[0], 3 * num_points);
		// Compute d1
		// d1 = E1 * x1_transformed
		m = 3, k = 3, n = num_points;
		kernels::gpu_blas_mmul_batched(d_E_canidate, x1_transformed, d1, m, k, n, m*k, 0, m* n, ransac_iter, handle);
		// }
		// Now calculate x2_transformed, n2 and d2 {
		m = 3, k = 3, n = num_points;
		kernels::gpu_blas_mmul_batched(d_E_canidate, X[1], x2_transformed, m, k, n, m*k, 0, m* n, ransac_iter, handle);
		//Compute n2
		m = num_points, k = 3, n = 3; // these probably need to change because we need to transpose X[0]
		kernels::gpu_blas_mmul_transpose_batched(X[0], d_E_canidate, n2, m, k, n, 0, 3 * 3, m * n, ransac_iter, handle); // transpose X[0]
		blocks = ceil((3 * num_points + cuda_block_size - 1) / cuda_block_size);
		kernels::element_wise_mult << <blocks, cuda_block_size >> > (n2, X[1], 3 * num_points);
		// Compute d2
		m = 3, k = 3, n = num_points;
		kernels::gpu_blas_mmul_batched(d_E_canidate, x2_transformed, d2, m, k, n, m*k, 0, m* n, ransac_iter, handle);
		// }
		// Now calculate the residual per canidate E{
		float *norm_n1, *norm_n2, *norm_d1, *norm_d2;
		int *inliers;
		int size = num_points * ransac_iter;
		hipMalloc((void**)&norm_n1, size * sizeof(float));
		hipMalloc((void**)&norm_n2, size * sizeof(float));
		hipMalloc((void**)&norm_d1, size * sizeof(float));
		hipMalloc((void**)&norm_d2, size * sizeof(float));
		hipMalloc((void**)&inliers, ransac_iter * sizeof(int));
		blocks = ceil((num_points * ransac_iter + cuda_block_size - 1) / cuda_block_size);
		kernels::vecnorm << <blocks, cuda_block_size >> > (n1, norm_n1, 3, size, 1, 2);
		kernels::vecnorm << <blocks, cuda_block_size >> > (n2, norm_n2, 3, size, 1, 2);

		kernels::vecnorm << <blocks, cuda_block_size >> > (d1, norm_d1, 3, size, 2, 2);
		kernels::vecnorm << <blocks, cuda_block_size >> > (d1, norm_d1, 3, size, 2, 2);

		kernels::element_wise_div << <blocks, cuda_block_size >> > (norm_n1, norm_d1, size);
		kernels::element_wise_div << <blocks, cuda_block_size >> > (norm_n2, norm_d2, size);
		// We now have the residuals in norm_n1
		kernels::element_wise_sum << <blocks, cuda_block_size >> > (norm_n1, norm_n2, size);
		// Calculate inliers per cell
		blocks = ceil((ransac_iter + cuda_block_size - 1) / cuda_block_size);
		kernels::threshold_count << <blocks, cuda_block_size >> > (norm_n1, inliers, num_points, ransac_iter, 1e-5); // tested
		//}
		// Not sure if we should free
		hipFree(n1);
		hipFree(n2);
		hipFree(d1);
		hipFree(d2);
		hipFree(x1_transformed);
		hipFree(x2_transformed);
		// Free the norms!!!
		hipFree(norm_n1);
		hipFree(norm_n2);
		hipFree(norm_d1);
		hipFree(norm_d2);
		// 100% free
		hipFree(d_E1);
		return inliers;
	}

	void Image_pair::computePoseCanidates() {
		// Tested
		float E[9];// = { -0.211 , -0.798 , -0.561, -0.967 , 0.252  , 0.009, 0.046  , 0.047  , 0.039 }; // TODO remove this once testing is done
		hipMemcpy(E, d_E, 3 * 3 * sizeof(float), hipMemcpyDeviceToHost);
		float u[9], d[9], v[9], tmp[9];
		svd(E, u, d, v); // v is not transposed
		multABt(u, v, tmp); // u * v'
		if (det(tmp) < 0)
			neg(v);
		float *d_u, *d_v;
		d_u = kernels::cuda_alloc_copy(u, 3 * 3);
		d_v = kernels::cuda_alloc_copy(v, 3 * 3);
		kernels::canidate_kernels << <1, 32 >> > (d_P, d_u, d_v);
		hipFree(d_u);
		hipFree(d_v);
	}

	void Image_pair::choosePose() {
		////Debugging{
		//	float *d_P_debugging;
		//	float x[] = { 1,2,1,2,
		//				2,1,2,1,
		//				1,1,2,2,
		//				1,2,2,1,
		//				
		//				1,1,2,2,
		//				1,2,1,2,
		//				2,1,2,1,
		//				1,2,2,1,
		//		
		//				1,2,1,2,
		//				1,1,2,2,
		//				2,1,2,1,
		//				1,2,2,1,
		//				
		//				1,2,2,1,
		//				1,2,1,2,
		//				2,1,2,1,
		//				1,1,2,2};
		//	d_P_debugging = kernels::cuda_alloc_copy(x, 4 * 4 * 4);
		////}
		// take 1 point and figure out if it is in front of the camera or behind
		float P1[16] = { 1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1 }; // I(4)
		float *d_P1 = kernels::cuda_alloc_copy(P1, 16);
		float *d_A, *d_u, *d_d, *d_vt;
		hipMalloc((void **)&d_A, 4 * 4 * 4 * sizeof(float));
		hipMalloc((void **)&d_u, 4 * 4 * 4 * sizeof(float));
		hipMalloc((void **)&d_d, 4 * 4 * sizeof(float));
		hipMalloc((void **)&d_vt, 4 * 4 * 4 * sizeof(float));
		// Create A

		dim3 blocks(1, 1);
		dim3 block_sizes(4, 2);
		kernels::compute_linear_triangulation_A << <blocks, block_sizes >> > (d_A, X[0], X[1], 4, num_points, d_P1, d_P, -1, true);
		kernels::print3DSlice(d_A, 4, 4, 0, 4, "d_A");
		// We only care about V
		float *d_d1, *d_d2; // 3x4 batched
		hipMalloc((void **)&d_d1, 4 * 4 * sizeof(float));
		hipMalloc((void **)&d_d2, 4 * 4 * sizeof(float));
		// Assumes V isnt transposed, we need to take the last row
		// svd(d_A, d_u, d_d, d_v, 4 batches)
		checkCUDAErrorWithLine("Before SVD");
		int *d_info = NULL;
		hipMalloc((void**)&d_info, 4 * sizeof(int));
		kernels::svd_square(d_A, d_vt, d_d, d_u, 4, 4, 4, d_info, cusolverH, stream, gesvdj_params);
		checkCUDAErrorWithLine("SVD");
		kernels::normalize_pt_kernal <<<1, 4 >> > (d_vt, d_d1, 4);
		kernels::printMatrix(d_d1, 4, 4, 4, "d1");

		float val_d1, val_d2;
		P_ind = 0;
		for (int i = 0; i < 4; i++) { // batched doesn't work for inverse + it is only 4, 4x4 matrices, should be easy
			kernels::invert(d_P + i * 4 * 4, d_P + i * 4 * 4, 4, 1, handle);
			int m = 4, k = 4, n = 4;
			kernels::gpu_blas_mmul(d_P + i * 4 * 4, d_d1, d_d2, m, k, n, handle);
			kernels::print3DSlice(d_P + i * 4 * 4, 4, 4, 0, 4, "d_P2_inv");
			kernels::printMatrix(d_d2, 4, 4, 4, "d2");
			// Do the final testing on the host
			hipMemcpy(&val_d1, &(d_d1[access2(2, i, 4)]), sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(&val_d2, &(d_d2[access2(2, i, 4)]), sizeof(float), hipMemcpyDeviceToHost);
			// Now we do the final check on the cpu as well, because it is the same ease
			if (val_d1 > 0 && val_d2 > 0)
				P_ind = i;
		}
		hipFree(d_P1);
		hipFree(d_A);
		hipFree(d_u);
		hipFree(d_d);
		hipFree(d_vt);
		hipFree(d_d1);
		hipFree(d_d2);
		hipFree(d_info);
	}

	void Image_pair::linear_triangulation() {
		// Similar to choosePose, except we know the pose we want so we don't need to do the later half of the computations
		// Tested
		float P1[16] = { 1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1 }; // I(4)
		float *d_P1 = kernels::cuda_alloc_copy(P1, 16);
		float *d_A, *d_u, *d_d, *d_vt;
		hipMalloc((void **)&d_A, 4 * 4 * num_points * sizeof(float));
		hipMalloc((void **)&d_u, 4 * 4 * num_points * sizeof(float));
		hipMalloc((void **)&d_d, 4 * num_points * sizeof(float));
		hipMalloc((void **)&d_vt, 4 * 4 * num_points * sizeof(float));
		// Create A

		dim3 grids(ceil((num_points * 2 + cuda_block_size - 1) / cuda_block_size), 1);
		dim3 block_sizes(cuda_block_size/2, 2);
		kernels::compute_linear_triangulation_A << <grids, block_sizes >> > (d_A, X[0], X[1], num_points, num_points, d_P1, d_P, P_ind, false); 
		checkCUDAError("A computation error");
		kernels::print3DSlice(d_A, 4, 4, 0, 4, "A[0]");
		// Assumes V isnt transposed, we need to take the last column
		int *d_info = NULL;
		hipMalloc((void**)&d_info, 4 * sizeof(int));
		kernels::svd_square(d_A, d_vt, d_d, d_u, 4, 4, num_points, d_info, cusolverH, stream, gesvdj_params);
		checkCUDAError("SVD error");
		kernels::print3DSlice(d_d, 4, 4, 0, 4, "d_vt[0]");
		dim3 grids2(ceil((num_points + cuda_block_size - 1) / cuda_block_size), 1);
		dim3 block_sizes2(cuda_block_size, 4);
		// Normalize by using the last row of v'
		kernels::normalize_pt_kernal <<<grids2, block_sizes2 >> > (d_vt, d_final_points, num_points);  
		kernels::printMatrix(d_final_points, 3, num_points, 5, "Transformed points");
		hipFree(d_P1);
		hipFree(d_A);
		hipFree(d_u);
		hipFree(d_d);
		hipFree(d_vt);
		hipFree(d_info);

	}

	Image_pair::~Image_pair() {
		hipFree(d_K);
		hipFree(d_K_inv);
		// Free vector points
		for (auto x : X)
			hipFree(x);
		for (auto x : U)
			hipFree(x);
		hipFree(d_final_points);
		// E
		hipFree(d_P);
		hipFree(d_E);
		hipblasDestroy(handle);
	}
	////////////////////////////////////////
	////////////////////////////////////////
	__host__ __device__ unsigned int hash(unsigned int a) {
		a = (a + 0x7ed55d16) + (a << 12);
		a = (a ^ 0xc761c23c) ^ (a >> 19);
		a = (a + 0x165667b1) + (a << 5);
		a = (a + 0xd3a2646c) ^ (a << 9);
		a = (a + 0xfd7046c5) + (a << 3);
		a = (a ^ 0xb55a4f09) ^ (a >> 16);
		return a;
	}

	
	void Image_pair::copyBoidsToVBO(float *vbodptr_positions) {
		dim3 fullBlocksPerGrid((num_points + cuda_block_size - 1) / cuda_block_size);
		checkCUDAErrorWithLine("Not copyBoidsToVBO failed!");
		kernels::kernCopyPositionsToVBO << <fullBlocksPerGrid, cuda_block_size >> > (num_points, d_final_points, vbodptr_positions, 100);
		//kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_vel1, vbodptr_velocities, scene_scale);

		checkCUDAErrorWithLine("copyBoidsToVBO failed!");

		hipDeviceSynchronize();
	}


	////////////////////////////////////////
	///////			Testing       //////////
	////////////////////////////////////////
	void Image_pair::testBatchedmult() {
		// C(m,n) = A(m,k) * B(k,n)
		float A[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 0, 1, 2, 3, 4, 5, 6, 7, 8 };
		float B[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9 };
		float *d_A, *d_B, *d_C;
		// Alloc
		hipMalloc((void**)&d_A, 9 * 2 * sizeof(float));
		hipMalloc((void**)&d_B, 9 * sizeof(float));
		hipMalloc((void**)&d_C, 6 * 3 * sizeof(float));
		// Copy
		hipMemcpy(d_A, A, 9 * 2 * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_B, B, 9 * sizeof(float), hipMemcpyHostToDevice);

		int m = 3, k = 1, n = 3;
		int lda = m;
		int ldb = k;
		int ldc = m;
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		const float alf = 1; // gpu vs cpu
		const float bet = 0;
		const float *alpha = &alf;
		const float *beta = &bet;
		int sA = 3;
		int sB = 0;
		int sC = 3;
		int batches = 6;
		/*hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			m, n, k, alpha, d_A, lda, 9, d_B, ldb, 0,
			beta, d_C, ldc, 3, 2);*/
		hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, d_B, n, sB, d_A, k, sA, beta, d_C, n, sC, batches);
		hipblasDestroy(handle);
		kernels::printVector(d_C, 6 * 3, "C");
		// Free test stuff
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
	}
	void Image_pair::testSVD() {
		float b[4 * 4 * 2] = { 1,2,3,4,5,6,7,8,9,10,11,12,1,2,3,4,5,6,7,8,10,11,12,14 };
		float *d_b = kernels::cuda_alloc_copy(b, 4 * 4 * 2);
		kernels::printMatrix(d_b, 4, 4, 4, "b");
		float *d_VT = NULL; 
		float *d_S = NULL;
		float *d_U = NULL;
		int *d_info = NULL; 

		hipMalloc((void**)&d_VT, sizeof(float) * 4 * 4 * 2);
		hipMalloc((void**)&d_U, sizeof(float) * 4 * 4 * 2);
		hipMalloc((void**)&d_S, sizeof(float) * 4 * 2);
		hipMalloc((void**)&d_info, 4 * sizeof(int));
		kernels::svd_square(d_b, d_VT, d_S, d_U, 4, 4, 2, d_info, cusolverH, stream, gesvdj_params);
		kernels::printMatrix(d_VT, 4, 4, 4, "VT");
		kernels::printMatrix(d_S, 4, 2, 2, "S");
		kernels::printMatrix(d_U, 4, 4, 4, "U");
	}
	void Image_pair::testInverse() {
		// Conclusion, batched doesn't work
		float a[] = { 0.9649, 0.9572, 0.1419, 0.1576, 0.4854, 0.4218,0.9706, 0.8003, 0.9157,
						1, 2, 0, 0, 2, 0, 1, 2, 1 };
		float *d_A = kernels::cuda_alloc_copy(a, 18);
		float *d_b;
		hipMalloc((void **)&d_b, 18 * sizeof(float));
		kernels::invert(d_A + 9, d_b, 3, 1, handle);
		kernels::printMatrix(d_b, 3, 3, 3, "b[0]");
		kernels::print3DSlice(d_b, 3, 3, 1, 3, "b[1]");
		hipFree(d_A);
		hipFree(d_b);
	}
	void Image_pair::testThrust_max() {
		int a[] = { 1,2,3,4,5,6, 4, 1, 3 };
		int *d_A = kernels::cuda_alloc_copy<int>(a, 7);

		thrust::device_ptr<int> dv_in(d_A);
		auto iter = thrust::max_element(dv_in, dv_in + 6);

		unsigned int position = iter - dv_in;
		int max_val = *iter;
		hipFree(d_A);
		std::cout << "The maximum value is " << max_val << " at position " << position << std::endl;
	}
	void Image_pair::testBatchedmultTranspose() {
		// Verdict: Works!!!
		float A[] = { 1, 2, 3, 1,
					  4, 5, 6, 1,
					  7, 8, 9, 1,

					  0, 1, 2, 1,
					  3, 4, 5, 1,
					  6, 7, 8, 1 };
		float B[] = { 1, 2, 3,
					  4, 5, 6,
					  7, 8, 9 };
		float *d_A, *d_B, *d_C;
		hipMalloc((void**)& d_C, 4 * 3 * 2 * sizeof(float));
		d_A = kernels::cuda_alloc_copy(A, 4 * 3 * 2);
		d_B = kernels::cuda_alloc_copy(B, 9);
		kernels::gpu_blas_mmul_transpose_batched(d_A, d_B, d_C, 4, 3, 3, 4 * 3, 0, 4 * 3, 2, handle);
		kernels::print3DSlice(d_C, 4, 3, 0, 3, "d_C[0]");
		kernels::print3DSlice(d_C, 4, 3, 1, 3, "d_C[1]");
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
	}
	void Image_pair::testRow_extraction_kernel() {
		float data[9 * 9 * 9];
		for (int i = 0; i < 9 * 9 * 9; i++)
			data[i] = i;
		float *d_d = kernels::cuda_alloc_copy(data, 9 * 9 * 9);
		float *res;
		hipMalloc((void**)&res, 9 * 9 * sizeof(float));

		kernels::row_extraction_kernel << <1, 9 >> > (d_d, res, 9);
		kernels::printMatrix(res, 9, 9, 9, "res");
		hipFree(res);
		hipFree(d_d);
	}
	void Image_pair::testVecnorm() {
		float test[9] = { 1,2,3,4,5,6,7,8,9 };
		float *test_gpu = kernels::cuda_alloc_copy(test, 9 * sizeof(float));
		float *norm_n1;
		hipMalloc((void**)&norm_n1, 3 * sizeof(float));
		kernels::vecnorm << <1, 3 >> > (test_gpu, norm_n1, 3, 3, 2, 1);
		kernels::printMatrix(norm_n1, 3, 1, 1, "norm");
	}
}