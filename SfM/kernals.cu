#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "kernals.h"
#include <assert.h>
#include <hipblas.h>
#include <hiprand.h>
#include <memory>
#include <string>
#include <iostream>
#include <fstream>
#include <hipblas.h>
#include <CudaSift/cudaSift.h>
#include <hiprand.h>
#include <iomanip>      
#include <random>
#include <algorithm>

namespace Kernals {
#define enable_debug true
#define esp 1e-5
#define x_pos 0
#define y_pos 1
#define z_pos 2
#define access2(i, j, col) i*col + j
#define access3(i, j, k, row, col) k * row * col + i*col + j
	using Common::PerformanceTimer;
	PerformanceTimer& timer()
	{
		static PerformanceTimer timer;
		return timer;
	}
	////////////////////////////////////////
	///////			Debugging	  //////////
	////////////////////////////////////////
	template<typename T>
	void printVector(const T *a1, int n, string name) {
		if (!enable_debug)
			return;
		T *print_a = new T[n];
		cout << name.c_str() << endl;
		cout << "{" << endl;
		hipMemcpy(print_a, a1, n * sizeof(T), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++) {
			cout << "\t" << print_a[i] << endl;
		}
		cout << "}" << endl;
		delete[]print_a;
	}
	template<typename T>
	void printMatrix(const T*A, int row, int col, int print_col, const char* name)
	{
		/// Prints first and last print_col values of A if A is a 2d matrix
		if (!enable_debug)
			return;
		T *print_a = new T[col*row];
		hipMemcpy(print_a, A, row* col * sizeof(T), hipMemcpyDeviceToHost);
		cout << name << endl;
		cout << "{" << endl;
		for (int i = 0; i < row; i++) {
			for (int j = 0; j < col; j++) {
				if (j < print_col || j > col - print_col - 1) {
					T Areg = print_a[access2(i, j, col)];
					cout << std::fixed << setprecision(3) << "\t"  << Areg;
				}
				else if (j == print_col) {
					cout << "\t....";
				}
			}
			cout << endl;
		}
		cout << "}" << endl;
		delete[]print_a;
	}
	template<typename T>
	void print3DSlice(const T*A, int row, int col, int slice, int print_col, const char* name)
	{
		/// Prints first and last print_col values of A if A is a 2d matrix
		if (!enable_debug)
			return;
		T *print_a = new T[col*row];
		hipMemcpy(print_a, A, row* col * sizeof(T), hipMemcpyDeviceToHost);
		cout << name << endl;
		cout << "{" << endl;
		for (int i = 0; i < row; i++) {
			for (int j = 0; j < col; j++) {
				if (j < print_col || j > col - print_col - 1) {
					T Areg = print_a[access3(i, j, slice, row, col)];
					cout << std::setw(7) << setprecision(3) << "\t\t" << Areg;
				}
				else if (j == print_col) {
					cout << "\t....";
				}
			}
			cout << endl;
		}
		cout << "}" << endl;
		delete[]print_a;
	}
	//////////////////////////////
	/*		Kernals			*/
	//////////////////////////////
	// C(m,n) = A(m,k) * B(k,n)
	// lda = k (if transposed)
	// ldb = n (if we transpose)
	// ldb = n (if we transpose)
	void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n, bool trans_flag_a, bool trans_flag_b) {
		int lda, ldb, ldc;
		lda = (!trans_flag_a) ? m : k;
		ldb = (!trans_flag_b) ? k : n;
		ldc = m;
		const float alf = 1; // gpu vs cpu
		const float bet = 0;
		const float *alpha = &alf;
		const float *beta = &bet;
		// Do the actual multiplication
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSgemm(handle, (hipblasOperation_t)trans_flag_a, (hipblasOperation_t)trans_flag_b, n, m, k, alpha, B, n, A, k, beta, C, n);
		hipblasDestroy(handle);
	}
	
	__global__ 
	void kron_kernal(float*d1, float*d2, float *A, int *indices, const int ransac_iterations, int num_points) {
		const int index = blockIdx.x*blockDim.x + threadIdx.x;
		const int A_row = 8;
		const int A_col = 9;
		
		if (access3(A_row - 1, A_col - 1, index, A_row, A_col) > ransac_iterations * A_row * A_col)
			return;
#pragma unroll
		for (int i = 0; i < A_row; i++) {
			// begin
			A[access3(i, 0, index, A_row, A_col)] = d1[access2(x_pos, indices[index + i], num_points)] * d2[access2(x_pos, indices[index + i], num_points)];
			A[access3(i, 1, index, A_row, A_col)] = d1[access2(x_pos, indices[index + i], num_points)] * d2[access2(y_pos, indices[index + i], num_points)];
			A[access3(i, 2, index, A_row, A_col)] = d1[access2(x_pos, indices[index + i], num_points)] * d2[access2(z_pos, indices[index + i], num_points)];
			// second												  
			A[access3(i, 3, index, A_row, A_col)] = d1[access2(y_pos, indices[index + i], num_points)] * d2[access2(x_pos, indices[index + i], num_points)];
			A[access3(i, 4, index, A_row, A_col)] = d1[access2(y_pos, indices[index + i], num_points)] * d2[access2(y_pos, indices[index + i], num_points)];
			A[access3(i, 5, index, A_row, A_col)] = d1[access2(y_pos, indices[index + i], num_points)] * d2[access2(z_pos, indices[index + i], num_points)];
			//third													  
			A[access3(i, 6, index, A_row, A_col)] = d1[access2(z_pos, indices[index + i], num_points)] * d2[access2(x_pos, indices[index + i], num_points)];
			A[access3(i, 7, index, A_row, A_col)] = d1[access2(z_pos, indices[index + i], num_points)] * d2[access2(y_pos, indices[index + i], num_points)];
			A[access3(i, 8, index, A_row, A_col)] = d1[access2(z_pos, indices[index + i], num_points)] * d2[access2(z_pos, indices[index + i], num_points)];
		}
	}

	__global__
	void copy_point(SiftPoint* data, int numPoints, float *U1, float *U2) {
		const int index_col = blockIdx.x*blockDim.x + threadIdx.x; // col is x to prevent warp divergence as much as possible in this naive implementation
		const int index_row = blockIdx.y*blockDim.y + threadIdx.y;
		if (index_row >= 3 || index_col >= numPoints)
			return;
		if (!index_row) {
			U1[access2(index_row, index_col, numPoints)] = data[index_col].xpos;
			U2[access2(index_row, index_col, numPoints)] = data[index_col].match_xpos;
		}
		else if (index_row == 1) {
			U1[access2(index_row, index_col, numPoints)] = data[index_col].ypos;
			U2[access2(index_row, index_col, numPoints)] = data[index_col].match_ypos;
		}
		else {
			U1[access2(index_row, index_col, numPoints)] = 1;
			U2[access2(index_row, index_col, numPoints)] = 1;
		}
	}
	
	__global__ 
	void normalizeE(float *E, int ransac_iterations) {
		const int index = blockIdx.x*blockDim.x + threadIdx.x;
		//svd(E);
	}
}

namespace SfM {
	Image_pair::Image_pair(float k[9], float k_inv[9], int image_count, int num_points) :image_count(image_count), num_points(num_points){ // num_points should be a array if we want to deal with more than 2 images
		hipMalloc((void**)&d_K, 3 * 3 * sizeof(float));
		checkCUDAErrorWithLine("Malloc failed!");
		hipMalloc((void**)&d_K_inv, 3 * 3 * sizeof(float));
		checkCUDAErrorWithLine("Malloc failed!");
		hipMemcpy(d_K, k, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_K_inv, k_inv, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
		// Allocate Point space
		float *d_u, *d_x;
		for (int i = 0; i < image_count; i++) {
			hipMalloc((void**)&d_u, 3 * num_points * sizeof(float));
			U.push_back(d_u);
			hipMalloc((void**)&d_x, 3 * num_points * sizeof(float));
			X.push_back(d_x);
		}
		// SVD handles
	}
	void Image_pair::estimateE() {
		const int ransac_count = floor(num_points/8);
		// Create random order of points (on cpu using std::shuffle)
		int *indices = new int[num_points];
		int *d_indices;
		hipMalloc((void **)&d_indices, num_points * sizeof(int));
		for (int i = 0; i < num_points; indices[i] = i, i++);
		// Shufle data
		std::random_device rd;
		std::mt19937 g(rd());
		shuffle(indices, indices + num_points, g);
		// Copy data to gpu
		hipMemcpy(d_indices, indices, num_points * sizeof(int), hipMemcpyHostToDevice);
		// Calculate all kron products correctly
		float *d_A;
		hipMalloc((void **)&d_A, 8 * 9 * ransac_count);
		int grids = ceil((ransac_count + cuda_block_size - 1) / cuda_block_size);
		Kernals::kron_kernal<<<grids, cuda_block_size >>>(X[0], X[1], d_A, d_indices, ransac_count, num_points);
		checkCUDAErrorWithLine("Kron failed!");
		Kernals::printMatrix(X[0], 3, num_points, 3, "Kron X[0]");
		Kernals::printMatrix(X[1], 3, num_points, 3, "Kron X[1]");
		Kernals::print3DSlice(d_A, 8, 9, 0, 9, "First Kron product");
		// Calculate batch SVD

		// Calculate target E's
		// Calculate number of inliers for each E
		// Pick best E
		// Free stuff
		hipFree(d_A);
		hipFree(d_indices);
		free(indices);
	}
	void Image_pair::FillXU(SiftPoint *data) {
		Kernals::printMatrix(d_K_inv, 3, 3, 3, "K inv");
		// Fill U
		dim3 grids(ceil((num_points + cuda_block_size - 1) / cuda_block_size), 1);
		dim3 blocks(cuda_block_size, 3);
		Kernals::copy_point << <grids, blocks >> > (data, num_points, U[0], U[1]);
		Kernals::printMatrix(U[0], 3, num_points, 5, "U[0]");
		// Fill X using X = inv(K) * U
		Kernals::gpu_blas_mmul(d_K_inv, U[0], X[0], 3, 3, num_points, false, false);
		Kernals::gpu_blas_mmul(d_K_inv, U[1], X[1], 3, 3, num_points, false, false);
		Kernals::printMatrix(X[0], 3, num_points, 5, "X[0]");
	}
	
}