#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "kernals.h"
#include <assert.h>
#include <hipblas.h>
#include <hiprand.h>
#include <memory>
#include <string>
#include <iostream>
#include <fstream>
#include <hipblas.h>
#include <CudaSift/cudaSift.h>
#include <hiprand.h>
#include <iomanip>      
#include <random>
#include <algorithm>

namespace Kernals {
#define enable_debug true
#define esp 1e-5
#define x_pos 0
#define y_pos 1
#define z_pos 2
#define access2(i, j, col) i*col + j
#define access3(i, j, k, row, col) k * row * col + i*col + j
	using Common::PerformanceTimer;
	PerformanceTimer& timer()
	{
		static PerformanceTimer timer;
		return timer;
	}
	////////////////////////////////////////
	///////			Debugging	  //////////
	////////////////////////////////////////
	template<typename T>
	void printVector(const T *a1, int n, string name) {
		if (!enable_debug)
			return;
		T *print_a = new T[n];
		cout << name.c_str() << endl;
		cout << "{" << endl;
		hipMemcpy(print_a, a1, n * sizeof(T), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++) {
			cout << "\t" << print_a[i] << endl;
		}
		cout << "}" << endl;
		delete[]print_a;
	}
	template<typename T>
	void printMatrix(const T*A, int row, int col, int print_col, const char* name)
	{
		/// Prints first and last print_col values of A if A is a 2d matrix
		if (!enable_debug)
			return;
		T *print_a = new T[col*row];
		hipMemcpy(print_a, A, row* col * sizeof(T), hipMemcpyDeviceToHost);
		cout << name << endl;
		cout << "{" << endl;
		for (int i = 0; i < row; i++) {
			for (int j = 0; j < col; j++) {
				if (j < print_col || j > col - print_col - 1) {
					T Areg = print_a[access2(i, j, col)];
					cout << std::fixed << setprecision(3) << "\t" << Areg;
				}
				else if (j == print_col) {
					cout << "\t....";
				}
			}
			cout << endl;
		}
		cout << "}" << endl;
		delete[]print_a;
	}
	template<typename T>
	void print3DSlice(const T*A, int row, int col, int slice, int print_col, const char* name)
	{
		/// reads 
		if (!enable_debug)
			return;
		T *print_a = new T[col * row];
		hipMemcpy(print_a, A + row * col * slice, row * col * sizeof(T), hipMemcpyDeviceToHost);
		cout << name << endl;
		cout << "{" << endl;
		for (int i = 0; i < row; i++) {
			for (int j = 0; j < col; j++) {
				if (j < print_col || j > col - print_col - 1) {
					T Areg = print_a[access2(i, j, col)];
					cout << std::setw(7) << setprecision(3) << "\t" << Areg;
				}
				else if (j == print_col) {
					cout << "\t....";
				}
			}
			cout << endl;
		}
		cout << "}" << endl;
		delete[]print_a;
	}
	//////////////////////////////
	/*		Kernals			*/
	//////////////////////////////
	// C(m,n) = A(m,k) * B(k,n)
	// lda = k (if transposed)
	// ldb = n (if we transpose)
	// ldb = n (if we transpose)
	void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n, bool trans_flag_a, bool trans_flag_b) {
		int lda, ldb, ldc;
		lda = (!trans_flag_a) ? m : k;
		ldb = (!trans_flag_b) ? k : n;
		ldc = m;
		const float alf = 1; // gpu vs cpu
		const float bet = 0;
		const float *alpha = &alf;
		const float *beta = &bet;
		// Do the actual multiplication
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSgemm(handle, (hipblasOperation_t)trans_flag_a, (hipblasOperation_t)trans_flag_b, n, m, k, alpha, B, n, A, k, beta, C, n);
		hipblasDestroy(handle);
	}

	__global__
		void kron_kernal(float*d1, float*d2, float *A, int *indices, const int ransac_iterations, int num_points) {
		const int index = blockIdx.x*blockDim.x + threadIdx.x;
		const int A_row = 8;
		const int A_col = 9;

		if (index > ransac_iterations)
			return;
#pragma unroll
		for (int i = 0; i < A_row; i++) {
			// begin
			A[access3(i, 0, index, A_row, A_col)] = d1[access2(x_pos, indices[index * A_row + i], num_points)] * d2[access2(x_pos, indices[index * A_row + i], num_points)];
			A[access3(i, 1, index, A_row, A_col)] = d1[access2(x_pos, indices[index * A_row + i], num_points)] * d2[access2(y_pos, indices[index * A_row + i], num_points)];
			A[access3(i, 2, index, A_row, A_col)] = d1[access2(x_pos, indices[index * A_row + i], num_points)] * d2[access2(z_pos, indices[index * A_row + i], num_points)];
			// second												  			    
			A[access3(i, 3, index, A_row, A_col)] = d1[access2(y_pos, indices[index * A_row + i], num_points)] * d2[access2(x_pos, indices[index * A_row + i], num_points)];
			A[access3(i, 4, index, A_row, A_col)] = d1[access2(y_pos, indices[index * A_row + i], num_points)] * d2[access2(y_pos, indices[index * A_row + i], num_points)];
			A[access3(i, 5, index, A_row, A_col)] = d1[access2(y_pos, indices[index * A_row + i], num_points)] * d2[access2(z_pos, indices[index * A_row + i], num_points)];
			//third													  			    
			A[access3(i, 6, index, A_row, A_col)] = d1[access2(z_pos, indices[index * A_row + i], num_points)] * d2[access2(x_pos, indices[index * A_row + i], num_points)];
			A[access3(i, 7, index, A_row, A_col)] = d1[access2(z_pos, indices[index * A_row + i], num_points)] * d2[access2(y_pos, indices[index * A_row + i], num_points)];
			A[access3(i, 8, index, A_row, A_col)] = d1[access2(z_pos, indices[index * A_row + i], num_points)] * d2[access2(z_pos, indices[index * A_row + i], num_points)];
		}
	}

	__global__
		void copy_point(SiftPoint* data, int numPoints, float *U1, float *U2) {
		const int index_col = blockIdx.x*blockDim.x + threadIdx.x; // col is x to prevent warp divergence as much as possible in this naive implementation
		const int index_row = blockIdx.y*blockDim.y + threadIdx.y;
		if (index_row >= 3 || index_col >= numPoints)
			return;
		if (!index_row) {
			U1[access2(index_row, index_col, numPoints)] = data[index_col].xpos;
			U2[access2(index_row, index_col, numPoints)] = data[index_col].match_xpos;
		}
		else if (index_row == 1) {
			U1[access2(index_row, index_col, numPoints)] = data[index_col].ypos;
			U2[access2(index_row, index_col, numPoints)] = data[index_col].match_ypos;
		}
		else {
			U1[access2(index_row, index_col, numPoints)] = 1;
			U2[access2(index_row, index_col, numPoints)] = 1;
		}
	}

	__global__
		void normalizeE(float *E, int ransac_iterations) {
		const int index = blockIdx.x*blockDim.x + threadIdx.x;
		if (index > ransac_iterations)
			return;
		float u[9], d[9], v[9];
		svd(&(E[access3(0, 0, index, 3, 3)]), u, d, v); // find correct E
		d[access2(2, 2, 3)] = 0;
		d[access2(1, 1, 3)] = 1;
		d[access2(0, 0, 3)] = 1;
		// E = U * D * V'
		float tmp_u[9];
		multAB(u, d, tmp_u);
		multABt(tmp_u, v, &(E[access3(0, 0, index, 3, 3)]));
	}
}

namespace SfM {

	Image_pair::Image_pair(float k[9], float k_inv[9], int image_count, int num_points) :image_count(image_count), num_points(num_points) { // num_points should be a array if we want to deal with more than 2 images
		hipMalloc((void**)&d_K, 3 * 3 * sizeof(float));
		checkCUDAErrorWithLine("Malloc failed!");
		hipMalloc((void**)&d_K_inv, 3 * 3 * sizeof(float));
		checkCUDAErrorWithLine("Malloc failed!");
		hipMemcpy(d_K, k, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_K_inv, k_inv, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
		// Allocate Point space
		float *d_u, *d_x;
		for (int i = 0; i < image_count; i++) {
			hipMalloc((void**)&d_u, 3 * num_points * sizeof(float));
			U.push_back(d_u);
			hipMalloc((void**)&d_x, 3 * num_points * sizeof(float));
			X.push_back(d_x);
		}
		// E
		hipMalloc((void **)&d_E, 3 * 3 * sizeof(float));
		// Canidate R, T
		float *d_tmp;
		for (int i = 0; i < 2; i++) {
			hipMalloc((void **)&d_tmp, 3 * 3 * sizeof(float));
			d_R.push_back(d_tmp);
			hipMalloc((void **)&d_tmp, 3 * sizeof(float));
			d_T.push_back(d_tmp);
		}
	}
	void Image_pair::estimateE() {
		const int ransac_count = floor(num_points / 8);
		// Create random order of points (on cpu using std::shuffle)
		int *indices = new int[num_points];
		int *d_indices;
		hipMalloc((void **)&d_indices, num_points * sizeof(int));
		for (int i = 0; i < num_points; indices[i] = i, i++);
		// Shufle data
		std::random_device rd;
		std::mt19937 g(rd());
		//shuffle(indices, indices + num_points, g);
		// Copy data to gpu
		hipMemcpy(d_indices, indices, num_points * sizeof(int), hipMemcpyHostToDevice);
		// Calculate all kron products correctly
		float *d_A;
		hipMalloc((void **)&d_A, 8 * 9 * ransac_count * sizeof(float));
		checkCUDAErrorWithLine("A malloc failed!");
		int grids = ceil((ransac_count + cuda_block_size - 1) / cuda_block_size);
		Kernals::kron_kernal << <grids, cuda_block_size >> > (X[0], X[1], d_A, d_indices, ransac_count, num_points);
		checkCUDAErrorWithLine("Kron failed!");
		Kernals::printMatrix(X[0], 3, num_points, 9, "Kron X[0]");
		Kernals::printMatrix(X[1], 3, num_points, 9, "Kron X[1]");
		Kernals::print3DSlice(d_A, 8, 9, ransac_count - 1, 9, "Kron product");
		float *d_E_canidate;
		hipMalloc((void **)&d_E_canidate, 3 * 3 * ransac_count * sizeof(float));
		// Calculate batch SVD

		// Last column of V becomes E
		// Calculate target E's
		
		Kernals::normalizeE << <grids, cuda_block_size >> > (d_E_canidate, ransac_count);
		// Calculate number of inliers for each E

		// Pick best E and allocate d_E and E

		// Free stuff
		hipFree(d_A);
		hipFree(d_indices);
		free(indices);
	}

	void Image_pair::fillXU(SiftPoint *data) {
		Kernals::printMatrix(d_K_inv, 3, 3, 3, "K inv");
		// Fill U
		dim3 grids(ceil((num_points + cuda_block_size - 1) / cuda_block_size), 1);
		dim3 blocks(cuda_block_size, 3);
		Kernals::copy_point << <grids, blocks >> > (data, num_points, U[0], U[1]);
		Kernals::printMatrix(U[0], 3, num_points, 5, "U[0]");
		// Fill X using X = inv(K) * U
		Kernals::gpu_blas_mmul(d_K_inv, U[0], X[0], 3, 3, num_points, false, false);
		Kernals::gpu_blas_mmul(d_K_inv, U[1], X[1], 3, 3, num_points, false, false);
		Kernals::printMatrix(X[0], 3, num_points, 5, "X[0]");
	}

	void Image_pair::computePoseCanidates() {
		// We will do all of this on the cpu because it is soo simple
		float E[9];
		hipMemcpy(E, d_E, 3 * 3 * sizeof(float), hipMemcpyDeviceToHost);
		float u[9], d[9], v[9];
		svd(E, u, d, v);
		float R_z[9] = { 0, -1, 0, 1, 0, 0, 0, 0, 1 };
		float canidate_T[2][3], canidate_R[2][9];
		// T1 = subset(U*R_z*d*U');
		float tmp_prod[9], tmp_prod2[9], T[9];
		for (int i = 0; i < 2; i++) {
			if (i) {// change signs for second iter
				R_z[1] = -R_z[1];
				R_z[3] = -R_z[3];
			}
			multAB(u, R_z, tmp_prod); //U * R_z 
			multAB(tmp_prod, d, tmp_prod2);// U * R_z * d
			multABt(tmp_prod2, u, T);
			canidate_T[i][0] = -T[access2(1, 2, 3)];
			canidate_T[i][1] = T[access2(0, 2, 3)];
			canidate_T[i][2] = -T[access2(0, 1, 3)];
			// R1
			multABt(u, R_z, tmp_prod);
			multABt(tmp_prod, v, canidate_R[i]);
			// Copy back to gpu
			hipMemcpy(d_R[i], canidate_R[i], 9 * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_T[i], canidate_T[i], 3 * sizeof(float), hipMemcpyHostToDevice);
		}
	}

	void Image_pair::testSVD() {
		float A[9] = { 1, 2, 3, 4, 5, 6, 7, 8, 9 };
		float u[9], d[9], v[9];
		/*hipMalloc((void **)&d_A, 9 * sizeof(float));
		hipMalloc((void **)&d_U, 9 * sizeof(float));
		hipMalloc((void **)&d_D, 9 * sizeof(float));
		hipMalloc((void **)&d_V, 9 * sizeof(float));
		hipMemcpy(d_A, A, 9 * sizeof(float), hipMemcpyHostToDevice);*/
		// Call svm
		/*svd(A[0], A[1], A[2], A[3], A[4], A[5], A[6], A[7], A[8],
			u[0], u[1], u[2], u[3], u[4], u[5], u[6], u[7], u[8],
			d[0], d[1], d[2], d[3], d[4], d[5], d[6], d[7], d[8],
			v[0], v[1], v[2], v[3], v[4], v[5], v[6], v[7], v[8]);*/
		svd(A, u, d, v);
		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 3; j++)
				cout << d[access2(i, j, 3)] << "\t";
			cout << endl;
		}
	}
}